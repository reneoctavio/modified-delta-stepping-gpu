/**
 * Modified Delta-Stepping Algorithm
 * Created on: Nov 13, 2015
 *
 * main.cu
 *
 * Author: Rene Octavio Queiroz Dias
 * License: GPLv3
 */

#include "delta_stepping_sssp.h"

int main(int argc, char* argv[]) {
	/*
	// Check correct number of arguments
	if ((argc != 3) && (argc != 5)) return 1;

	// Read graph
	////////////////////////////////////////////////////////////////////////////
    struct timeval time;
    gettimeofday(&time, NULL);
    double t1_read = time.tv_sec + (time.tv_usec / 1000000.0);

	DCsrMatrix d_graph;

	if 		(std::strcmp(argv[1], "d") == 0) { cusp::io::read_dimacs_file(d_graph, argv[2]); }
	else if (std::strcmp(argv[1], "m") == 0) { cusp::io::read_matrix_market_file(d_graph, argv[2]); }
	else 					 				 { printf("Invalid format.\n"); return 1; }
	d_graph.row_offsets.shrink_to_fit();
	d_graph.column_indices.shrink_to_fit();
	d_graph.values.shrink_to_fit();

	gettimeofday(&time, NULL);
    double t2_read = time.tv_sec + (time.tv_usec / 1000000.0);
    printf("\nRead data time: %.6lf seconds\n\n", t2_read - t1_read);
    ////////////////////////////////////////////////////////////////////////////

    // Calculate degree and average edge length
    ////////////////////////////////////////////////////////////////////////////
	DVector d_edges_count(d_graph.row_offsets.size());
	thrust::adjacent_difference(d_graph.row_offsets.begin(),
								d_graph.row_offsets.end(),
								d_edges_count.begin());
	d_edges_count.erase(d_edges_count.begin());

	const int k_avg_degree = thrust::reduce(d_edges_count.begin(), d_edges_count.end()) / d_edges_count.size();
	const int k_avg_edge_length = thrust::reduce(d_graph.values.begin(), d_graph.values.end()) / d_graph.values.size();

	d_edges_count.clear();
	d_edges_count.shrink_to_fit();
	////////////////////////////////////////////////////////////////////////////

	// Update Properties
	////////////////////////////////////////////////////////////////////////////
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int delta = prop.warpSize * k_avg_edge_length / k_avg_degree;
	const int k_delta = (delta > 0) ? delta : 1;
	delta = k_delta;
	////////////////////////////////////////////////////////////////////////////

	// Print properties
	std::cout << "Device: " << prop.name << std::endl;
	std::cout << "Warp Size: " << prop.warpSize << std::endl;
	std::cout << "Average degree: " << k_avg_degree << std::endl;
	std::cout << "Average edge length: " << k_avg_edge_length << std::endl;
	std::cout << "Calculated delta: " << k_delta << std::endl;

	// Create distance vector
	DVector d_distance(d_graph.num_rows);
	d_distance.shrink_to_fit();

	// STATS
	int sple = 1;
	int n_sple = 1;
#if STATS
	int num_samples = std::atoi(argv[4]);
	n_sple = num_samples;

	// Read deltas
	cusp::array1d<int, cusp::host_memory> deltas;
	cusp::io::read_matrix_market_file(deltas, argv[3]);
	thrust::sort(thrust::host, deltas.begin(), deltas.end());

	cusp::array2d<double, cusp::host_memory> samples_sssp(deltas.size(), num_samples);
	cusp::array1d<double, cusp::host_memory> samples_sep(deltas.size());
	cusp::array1d<double, cusp::host_memory> samples_sep_light_edg(deltas.size());
	cusp::array1d<double, cusp::host_memory> samples_sep_heavy_edg(deltas.size());

	for (int delta_idx = 0; delta_idx < deltas.size(); delta_idx++) {
		delta = deltas[delta_idx];
#endif
		// Separate graph
	    ////////////////////////////////////////////////////////////////////////////
	    gettimeofday(&time, NULL); double t1_sep = time.tv_sec + (time.tv_usec / 1000000.0);

		DCsrMatrix d_graph_light, d_graph_heavy;
		separate_graphs(&d_graph_light, &d_graph_heavy, &d_graph, delta);

		gettimeofday(&time, NULL); double t2_sep = time.tv_sec + (time.tv_usec / 1000000.0);
	    ////////////////////////////////////////////////////////////////////////////
		printf("\nDelta: %d, Separation Time: %.6lf seconds\n", delta, t2_sep - t1_sep);
		printf("\nDelta: %d, Light Edges: %d, Heavy Edges: %d\n", delta, (int)d_graph_light.num_entries, (int)d_graph_heavy.num_entries);
#if STATS
	    samples_sep[delta_idx] = t2_sep - t1_sep;
	    samples_sep_light_edg[delta_idx] = d_graph_light.num_entries;
	    samples_sep_heavy_edg[delta_idx] = d_graph_heavy.num_entries;

		for (int sample = 0; sample < num_samples; sample++) {
			sple = sample + 1;
#endif
			// Run SSSP
			////////////////////////////////////////////////////////////////////////////
			gettimeofday(&time, NULL); double t1 = time.tv_sec + (time.tv_usec / 1000000.0);

			//delta_stepping_gpu_sssp(&d_graph_light, &d_graph_heavy, &d_distance, delta, 0);
			delta_stepping_gpu_csr(&d_graph_light, &d_graph_heavy, &d_distance, delta, 0);
			gettimeofday(&time, NULL); double t2 = time.tv_sec + (time.tv_usec / 1000000.0);
			////////////////////////////////////////////////////////////////////////////
			printf("Delta: %d, Sample #: %d of %d, Computation Time: %.6lf seconds\n", delta, sple, n_sple, t2 - t1);
#if STATS
			samples_sssp(delta_idx, sample) = t2 - t1;
		}
	}
	// Write statistics
	cusp::io::write_matrix_market_file(deltas, "deltas.mtx");
	cusp::io::write_matrix_market_file(samples_sssp, "time_samples_sssp.mtx");
	cusp::io::write_matrix_market_file(samples_sep, "time_samples_sep.mtx");
	cusp::io::write_matrix_market_file(samples_sep_light_edg, "samples_sep_light_edg.mtx");
	cusp::io::write_matrix_market_file(samples_sep_heavy_edg, "samples_sep_heavy_edg.mtx");
#endif

    // Write distances
    cusp::io::write_matrix_market_file(d_distance, "distance.mtx");
    */

	ContextPtr context = CreateCudaDevice(0);

	HCsrMatrix h_graph, h_graph_light, h_graph_heavy;
	int delta = std::atoi(argv[3]);

	if 		(std::strcmp(argv[1], "d") == 0) { cusp::io::read_dimacs_file(h_graph, argv[2]); }
	else if (std::strcmp(argv[1], "m") == 0) { cusp::io::read_matrix_market_file(h_graph, argv[2]); }
	else 					 				 { printf("Invalid format.\n"); return 1; }

	separate_graphs_host(&h_graph_light, &h_graph_heavy, &h_graph, delta);

	delta_stepping_gpu_mpgu(*context, &h_graph_light, &h_graph_heavy, delta, 0);

	return 0;
}
